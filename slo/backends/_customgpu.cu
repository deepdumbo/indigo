#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_complex.h>

#include "_customgpu.h"

__global__
void cu_exw_csrmm_H(unsigned int M, unsigned int N, unsigned int K,
    hipFloatComplex alpha, hipFloatComplex *values,
    unsigned int *colInds, unsigned int *rowPtrs,
    hipFloatComplex *X, unsigned int ldx, hipFloatComplex beta,
    hipFloatComplex *Y, unsigned int ldy)
{
    int m = blockIdx.x*blockDim.x + threadIdx.x;
    if (m >= M)
        return;

    extern __shared__ hipFloatComplex x[];

    unsigned int ptrb = rowPtrs[m],
                 ptre = rowPtrs[m+1];

    if (ptrb == ptre)
        return;

    #pragma unroll
    for (unsigned int n = 0; n < N; n++)
        x[n] = X[m+n*M];

    for (unsigned int idx = ptrb; idx < ptre; idx++) {
        unsigned int k = colInds[idx];
        hipFloatComplex v = hipCmulf(alpha, hipConjf(values[idx]));

        #pragma unroll
        for (unsigned int n = 0; n < N; n++)
            Y[k+n*K] = hipCmulf(v, x[n]);
    }
}

extern "C"
void c_exw_csrmm_H(unsigned int M, unsigned int N, unsigned int K,
    hipFloatComplex alpha, hipFloatComplex *values,
    unsigned int *colInds, unsigned int *rowPtrs,
    hipFloatComplex *X, unsigned int ldx, hipFloatComplex beta,
    hipFloatComplex *Y, unsigned int ldy)
{
    // Y[:] *= beta
    hipblasCscal(K*N, beta, Y, 1);

    // Y[:] += alpha * AX
    int tpb = 128;
    int nb = (M+tpb-1)/tpb;
    int ns = N * sizeof(hipFloatComplex);
    cu_exw_csrmm_H<<<nb,tpb,ns>>>(M, N, K, alpha, values, colInds, rowPtrs, X, ldx, beta, Y, ldy);
}
