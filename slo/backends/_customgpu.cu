#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_complex.h>

#include "_customgpu.h"

__global__
void cu_exw_csrmm_H(unsigned int M, unsigned int N, unsigned int K,
    hipFloatComplex alpha, hipFloatComplex *values,
    unsigned int *colInds, unsigned int *rowPtrs,
    hipFloatComplex *X, unsigned int ldx, hipFloatComplex beta,
    hipFloatComplex *Y, unsigned int ldy)
{
    int m = blockIdx.x*blockDim.x + threadIdx.x;
    if (m >= M)
        return;

    for (unsigned int idx = rowPtrs[m]; idx < rowPtrs[m+1]; idx++) {
        unsigned int k = colInds[idx];
        hipFloatComplex v = hipCmulf(alpha, hipConjf(values[idx]));

        #pragma unroll
        for (unsigned int n = 0; n < N; n++)
            Y[k+n*K] = hipCmulf(v, X[m+n*M]);
    }
}

extern "C"
void c_exw_csrmm_H(unsigned int M, unsigned int N, unsigned int K,
    hipFloatComplex alpha, hipFloatComplex *values,
    unsigned int *colInds, unsigned int *rowPtrs,
    hipFloatComplex *X, unsigned int ldx, hipFloatComplex beta,
    hipFloatComplex *Y, unsigned int ldy)
{
    // Y[:] *= beta
    hipblasCscal(K*N, beta, Y, 1);

    // Y[:] += alpha * AX
    int tpb = 128;
    int nb = (M+tpb-1)/tpb;
    cu_exw_csrmm_H<<<nb,tpb>>>(M, N, K, alpha, values, colInds, rowPtrs, X, ldx, beta, Y, ldy);
}
